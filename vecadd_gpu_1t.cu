
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

// now this is a CUDA kernel function that is a device code which runs on the GPU
__global__
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
    {
        y[i] = x[i] + y[i];
    }
}

int main(void)
{
    int N = 1 << 28; // 256M elements

    
    /*
    float *x = new float[N];
    float *y = new float[N];
    */

    // allocate unified memory that's reachable from both the CPU and the GPU
    float *x, *y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // start timer
    std::chrono::time_point<std::chrono::high_resolution_clock> start_time = std::chrono::high_resolution_clock::now();

    // Run kernel on 1M elements on the CPU
    // add(N, x, y);

    // Run kernel on elements on the GPU
    add<<<1, 1>>>(N, x, y);

    // end timer
    std::chrono::time_point<std::chrono::high_resolution_clock> end_time = std::chrono::high_resolution_clock::now();

    // we have to wait for the GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // output runtime
    std::chrono::duration<double> elapsed = end_time - start_time;
    std::cout << "Elapsed time: " << elapsed.count() << " s\n";

    // Free memory
    // delete[] x;
    // delete[] y;
    hipFree(x);
    hipFree(y); // we "malloc'd" them with CUDA so it's now cudaFree

    return 0;
}
